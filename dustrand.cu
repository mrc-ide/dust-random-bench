#include "hip/hip_runtime.h"
// -*-c++-*-
#include <chrono>
#include <iostream>

#include "common.hpp"
#include <dust/random/random.hpp>

using rng_state_type = dust::random::xoshiro128plus_state;
using rng_int_type = rng_state_type::int_type;

template <typename T>
__device__
T get_rng(const typename T::int_type * data, size_t index, size_t n) {
  T ret;
  for (size_t i = 0, j = i; i < ret.size(); ++i, j += n) {
    ret[i] = data[j];
  }
  return ret;
}

template <typename T>
__device__
void set_rng(T& rng, typename T::int_type * data, size_t n) {
  for (size_t i = 0, j = i; i < rng.size(); ++i, j += n) {
    data[j] = rng[i];
  }
}

__global__
void sample_uniform(rng_int_type * rng_state_data,
                    float *draws, size_t n_threads, size_t n_draws) {
  const int dx = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_threads; i += dx) {
    auto rng_block = get_rng<rng_state_type>(rng_state_data, i, n_threads);

    float draw = 0;
    for (int j = 0; j < n_draws; ++j) {
      float new_draw = dust::random::random_real<float>(rng_block);
      draw += new_draw;
    }
    draws[i] = draw;

    set_rng(rng_block, rng_state_data, n_threads);
  }
}

__global__
void sample_normal(rng_int_type * rng_state_data,
                   float *draws, size_t n_threads, size_t n_draws) {
  const int dx = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_threads; i += dx) {
    auto rng_block = get_rng<rng_state_type>(rng_state_data, i, n_threads);

    float draw = 0;
    for (int j = 0; j < n_draws; ++j) {
      float new_draw = dust::random::normal<float>(rng_block, 0, 1);
      draw += new_draw;
    }
    draws[i] = draw;

    set_rng(rng_block, rng_state_data, n_threads);
  }
}

__global__
void sample_exponential(rng_int_type * rng_state_data,
                        float *draws, size_t n_threads, size_t n_draws) {
  const int dx = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_threads; i += dx) {
    auto rng_block = get_rng<rng_state_type>(rng_state_data, i, n_threads);

    float draw = 0;
    for (int j = 0; j < n_draws; ++j) {
      float new_draw = dust::random::exponential<float>(rng_block, 1);
      draw += new_draw;
    }
    draws[i] = draw;

    set_rng(rng_block, rng_state_data, n_threads);
  }
}

__global__
void sample_poisson(rng_int_type * rng_state_data,
                        float *draws, size_t n_threads, size_t n_draws) {
  const int dx = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_threads; i += dx) {
    auto rng_block = get_rng<rng_state_type>(rng_state_data, i, n_threads);

    float draw = 0;
    for (int j = 0; j < n_draws; ++j) {
      float new_draw = dust::random::poisson<float>(rng_block, 1);
      draw += new_draw;
    }
    draws[i] = draw;

    set_rng(rng_block, rng_state_data, n_threads);
  }
}

__global__
void sample_binomial(rng_int_type * rng_state_data,
                        float *draws, size_t n_threads, size_t n_draws) {
  const int dx = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_threads; i += dx) {
    auto rng_block = get_rng<rng_state_type>(rng_state_data, i, n_threads);

    float draw = 0;
    for (int j = 0; j < n_draws; ++j) {
      float new_draw = dust::random::binomial<float>(rng_block, 10, 0.3);
      draw += new_draw;
    }
    draws[i] = draw;

    set_rng(rng_block, rng_state_data, n_threads);
  }
}

void run(const char * distribution_name, size_t n_threads, size_t n_draws) {
  auto distribution_type = check_distribution(distribution_name);
  float* draws;
  CUDA_CALL(hipMalloc((void**)&draws, n_threads * sizeof(float)));

  const size_t blockSize = 128;
  const size_t blockCount = (n_threads + blockSize - 1) / blockSize;

  auto t0_setup = std::chrono::high_resolution_clock::now();

  // This is currently done in series on the cpu, and will be quite slow.
  // First, initialise all random number generators
  const int seed = 42;
  dust::random::prng<rng_state_type> rng(n_threads, seed);
  constexpr auto rng_len = rng_state_type::size();

  // Then create a vector of integers representing the underlying
  // random number state, interleaved.
  std::vector<rng_int_type> rng_interleaved(n_threads * rng_len);
  for (size_t i = 0; i < n_threads; ++i) {
    auto p = rng.state(i);
    for (size_t j = 0, at = i; j < rng_len; ++j, at += n_threads) {
      rng_interleaved[at] = p[j];
    }
  }

  rng_int_type* rng_state;
  const size_t len = n_threads * rng_len * sizeof(rng_int_type);
  CUDA_CALL(hipMalloc((void**)&rng_state, len));
  CUDA_CALL(hipMemcpy(rng_state, rng_interleaved.data(), len,
                       hipMemcpyDefault));
  auto t1_setup = std::chrono::high_resolution_clock::now();

  auto t0_sample = std::chrono::high_resolution_clock::now();
  switch(distribution_type) {
  case UNIFORM:
    sample_uniform<<<blockCount, blockSize>>>(rng_state, draws,
                                              n_threads, n_draws);
    break;
  case NORMAL:
    sample_normal<<<blockCount, blockSize>>>(rng_state, draws,
                                              n_threads, n_draws);
    break;
  case EXPONENTIAL:
    sample_exponential<<<blockCount, blockSize>>>(rng_state, draws,
                                                  n_threads, n_draws);
    break;
  case POISSON:
    sample_poisson<<<blockCount, blockSize>>>(rng_state, draws,
                                              n_threads, n_draws);
    break;
  case BINOMIAL:
    sample_binomial<<<blockCount, blockSize>>>(rng_state, draws,
                                               n_threads, n_draws);
    break;
  }
  CUDA_CALL(hipDeviceSynchronize());
  auto t1_sample = std::chrono::high_resolution_clock::now();

  std::chrono::duration<double> t_setup = t1_setup - t0_setup;
  std::chrono::duration<double> t_sample = t1_sample - t0_sample;

  std::cout <<
    "engine: dust" <<
    ", distribution: " << distribution_name <<
    ", n_threads: " << n_threads <<
    ", n_draws: " << n_draws <<
    ", t_setup: " << t_setup.count() <<
    ", t_sample: " << t_sample.count() <<
    std::endl;

  CUDA_CALL(hipFree(draws));
  CUDA_CALL(hipFree(rng_state));
}

int main(int argc, char *argv[]) {
  if (argc != 4) {
    std::cout << "Usage: dustrand <type> <n_threads> <n_draws>" << std::endl;
    return 1;
  }

  try {
    auto type_str = argv[1];
    auto n_threads = std::stoi(argv[2]);
    auto n_draws = std::stoi(argv[3]);
    run(type_str, n_threads, n_draws);
  } catch (const std::exception& e) {
    std::cout << "Error: " << e.what() << std::endl;
    return 1;
  }

  return 0;
}
