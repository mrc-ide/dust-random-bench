#include "hip/hip_runtime.h"
// -*-c++-*-

#include <chrono>
#include <iostream>
#include <sstream>

// This needs to be put into numeric.hpp, I think
#include <cfloat>

#define DEVICE __device__
#define HOST __host__
#define HOSTDEVICE __host__ __device__
#define KERNEL __global__
#define ALIGN(n) __align__(n)

#define __nv_exec_check_disable__ _Pragma("nv_exec_check_disable")

#ifdef __CUDA_ARCH__
#define CONSTANT __constant__
#define SYNCWARP __syncwarp();
#else
#define CONSTANT const
#define SYNCWARP
#endif

#include "common.hpp"
#include <dust/random/random.hpp>
#include "helper.hpp"

using rng_state_type = dust::random::xoshiro128plus_state;
using rng_int_type = rng_state_type::int_type;

__global__
void sample_uniform(rng_int_type * rng_state,
                    float *draws, const long nthreads, const int ndraws) {
  const int dx = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nthreads; i += dx) {
    interleaved<rng_int_type> p_rng(rng_state, static_cast<size_t>(i), static_cast<size_t>(nthreads));

    rng_state_type rng_block;
    for (size_t j = 0; j < rng_block.size(); j++) {
      rng_block.state[j] = p_rng[j];
    }

    float draw = 0;
    for (int j = 0; j < ndraws; ++j) {
      float new_draw = dust::random::random_real<float>(rng_block);
      draw += new_draw;
      // __syncwarp();
    }
    draws[i] = draw;

    // TODO: Tidy this up; could use put_rng_state in a bit I think?
    for (size_t j = 0; j < rng_block.size(); j++) {
      p_rng[j] = rng_block.state[j];
    }
  }
}

int main(int argc, char *argv[]) {
  using namespace std::chrono;

  if (argc != 3) {
    std::cout << "Usage: hiprand <nthreads> <ndraws>" << std::endl;
    return 1;
  }

  const long nthreads = std::stoi(argv[1]);
  const int ndraws = std::stoi(argv[2]);

  float* draws;
  CUDA_CALL(hipMalloc((void**)&draws, nthreads * sizeof(float)));

  const size_t blockSize = 128;
  const size_t blockCount = (nthreads + blockSize - 1) / blockSize;

  auto t0_setup = high_resolution_clock::now();

  // This is currently done in series on the cpu, and will be quite slow.

  // First, initialise all random number generators
  const int seed = 42;
  dust::random::prng<rng_state_type> rng(nthreads, seed);
  constexpr auto rng_len = rng_state_type::size();

  // Then create a vector of integers representing the underlying
  // random number state, interleaved.
  std::vector<rng_int_type> rng_interleaved(nthreads * rng_len);
  for (size_t i = 0; i < nthreads; ++i) {
    auto p = rng.state(i);
    for (size_t j = 0, at = i; j < rng_len; ++j, at += nthreads) {
      rng_interleaved[at] = p[j];
    }
  }

  device_array<rng_int_type> rng_state(nthreads * rng_len);
  rng_state.set_array(rng_interleaved);

  auto t1_setup = high_resolution_clock::now();

  auto t0_sample = high_resolution_clock::now();
  sample_uniform<<<blockCount, blockSize>>>(rng_state.data(), draws,
                                            nthreads, ndraws);
  CUDA_CALL(hipDeviceSynchronize());
  auto t1_sample = high_resolution_clock::now();

  auto t_setup = duration_cast<duration<double>>(t1_setup - t0_setup);
  auto t_sample = duration_cast<duration<double>>(t1_sample - t0_sample);

  std::cout << "nthreads: " << nthreads <<
    ", ndraws: " << ndraws <<
    ", t_setup: " << t_setup.count() <<
    ", t_sample: " << t_sample.count() <<
    std::endl;
}
