// -*-c++-*-
// See the nvidia docs:
// https://docs.nvidia.com/cuda/curand/device-api-overview.html#poisson-api-example

#include <chrono>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

__global__
void setup_kernel(hiprandState *state, const long nthreads) {
  const int dx = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nthreads; i += dx) {
    hiprand_init(1234, i, 0, &state[i]);
  }
}

__global__
void sample_uniform(hiprandState *state, float *draws,
                    const long nthreads, const int ndraws) {
  const int dx = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nthreads; i += dx) {
    hiprandState localState = state[i];
    float draw = 0;
    for (int j = 0; j < ndraws; ++j) {
      float new_draw = hiprand_uniform(&localState);
      draw += new_draw;
      //__syncwarp();
    }
    draws[i] = draw;
    state[i] = localState;
  }
}

int main(int argc, char *argv[]) {
  using namespace std::chrono;
  if (argc != 3) {
    std::cout << "Usage: hiprand <nthreads> <ndraws>" << std::endl;
    return 1;
  }

  const long nthreads = std::stoi(argv[1]);
  const int ndraws = std::stoi(argv[2]);

  hiprandState *devStates;
  float* draws;
  CUDA_CALL(hipMalloc((void**)&draws, nthreads * sizeof(float)));
  CUDA_CALL(hipMalloc((void **)&devStates, nthreads *
              sizeof(hiprandState)));

  const size_t blockSize = 128;
  const size_t blockCount = (nthreads + blockSize - 1) / blockSize;

  auto t0_setup = high_resolution_clock::now();
  setup_kernel<<<blockCount, blockSize>>>(devStates, nthreads);
  CUDA_CALL(hipDeviceSynchronize());
  auto t1_setup = high_resolution_clock::now();

  auto t0_sample = high_resolution_clock::now();
  sample_uniform<<<blockCount, blockSize>>>(devStates, draws, nthreads, ndraws);
  CUDA_CALL(hipDeviceSynchronize());
  auto t1_sample = high_resolution_clock::now();

  CUDA_CALL(hipFree(draws));
  CUDA_CALL(hipFree(devStates));

  auto t_setup = duration_cast<duration<double>>(t1_setup - t0_setup);
  auto t_sample = duration_cast<duration<double>>(t1_sample - t0_sample);

  std::cout << "nthreads: " << nthreads <<
    ", ndraws: " << ndraws <<
    ", t_setup: " << t_setup.count() <<
    ", t_sample: " << t_sample.count() <<
    std::endl;
}
