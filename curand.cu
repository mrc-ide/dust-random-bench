#include "hip/hip_runtime.h"
// -*-c++-*-
// See the nvidia docs:
// https://docs.nvidia.com/cuda/hiprand/device-api-overview.html#poisson-api-example

#include <chrono>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include "common.hpp"

__global__
void setup_kernel(hiprandState *state, const long nthreads) {
  const int dx = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nthreads; i += dx) {
    hiprand_init(1234, i, 0, &state[i]);
  }
}

__global__
void sample_uniform(hiprandState *state, float *draws,
                    const long nthreads, const int ndraws) {
  const int dx = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nthreads; i += dx) {
    hiprandState localState = state[i];
    float draw = 0;
    for (int j = 0; j < ndraws; ++j) {
      float new_draw = hiprand_uniform(&localState);
      draw += new_draw;
    }
    draws[i] = draw;
    state[i] = localState;
  }
}

__global__
void sample_normal(hiprandState *state, float *draws,
                   const long nthreads, const int ndraws) {
  const int dx = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nthreads; i += dx) {
    hiprandState localState = state[i];
    float draw = 0;
    for (int j = 0; j < ndraws; ++j) {
      float new_draw = hiprand_normal(&localState);
      draw += new_draw;
    }
    draws[i] = draw;
    state[i] = localState;
  }
}

__global__
void sample_poisson(hiprandState *state, float *draws,
                    const long nthreads, const int ndraws) {
  const int dx = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nthreads; i += dx) {
    hiprandState localState = state[i];
    float draw = 0;
    for (int j = 0; j < ndraws; ++j) {
      float new_draw = hiprand_poisson(&localState, 1);
      draw += new_draw;
    }
    draws[i] = draw;
    state[i] = localState;
  }
}

void run(const char * distribution_name, size_t nthreads, size_t ndraws) {
  auto distribution_type = check_distribution(distribution_name);

  hiprandState *devStates;
  float* draws;
  CUDA_CALL(hipMalloc((void**)&draws, nthreads * sizeof(float)));

  const size_t blockSize = 128;
  const size_t blockCount = (nthreads + blockSize - 1) / blockSize;

  auto t0_setup = std::chrono::high_resolution_clock::now();
  CUDA_CALL(hipMalloc((void **)&devStates, nthreads * sizeof(hiprandState)));

  setup_kernel<<<blockCount, blockSize>>>(devStates, nthreads);
  CUDA_CALL(hipDeviceSynchronize());
  auto t1_setup = std::chrono::high_resolution_clock::now();

  auto t0_sample = std::chrono::high_resolution_clock::now();
  switch (distribution_type) {
  case UNIFORM:
    sample_uniform<<<blockCount, blockSize>>>(devStates, draws, nthreads,
                                              ndraws);
    break;
  case NORMAL:
    sample_normal<<<blockCount, blockSize>>>(devStates, draws, nthreads,
                                             ndraws);
    break;
  case POISSON:
    sample_poisson<<<blockCount, blockSize>>>(devStates, draws, nthreads,
                                              ndraws);
    break;
  default:
    std::stringstream msg;
    msg << "Distribution not supported with hiprand: " << distribution_name;
    throw std::runtime_error(msg.str());
  }
  CUDA_CALL(hipDeviceSynchronize());
  auto t1_sample = std::chrono::high_resolution_clock::now();

  CUDA_CALL(hipFree(draws));
  CUDA_CALL(hipFree(devStates));

  std::chrono::duration<double> t_setup = t1_setup - t0_setup;
  std::chrono::duration<double> t_sample = t1_sample - t0_sample;

  std::cout <<
    "distribution: " << distribution_name <<
    ", nthreads: " << nthreads <<
    ", ndraws: " << ndraws <<
    ", t_setup: " << t_setup.count() <<
    ", t_sample: " << t_sample.count() <<
    std::endl;
}

int main(int argc, char *argv[]) {
  if (argc != 4) {
    std::cout << "Usage: hiprand <type> <nthreads> <ndraws>" << std::endl;
    return 1;
  }

  try {
    auto type_str = argv[1];
    const long nthreads = std::stoi(argv[2]);
    const int ndraws = std::stoi(argv[3]);
    run(type_str, nthreads, ndraws);
  } catch (const std::exception& e) {
    std::cout << "Error: " << e.what() << std::endl;
    return 1;
  }

  return 0;
}
